#include<stdio.h>
#include <hip/hip_runtime.h>
#define N 1200


char input_file[]= "linsys_1200.txt";
float a[N][N];
float b[N];



__global__ void add(float *d_a, float *d_b, int *d_i) {
	float temp;
	int i = *d_i;
	
	//BlockId: the equation
	int j = blockIdx.x + i + 1;
	
	//ThreadId: column
	int k = threadIdx.x + i;

	temp = d_a[j*N+i]/d_a[i*N+i];
	
	if (k==N)
		d_b[j] = d_b[j] - d_b[i]*temp;

	else
		d_a[j*N+k] = d_a[j*N+k] - d_a[i*N+k]*temp;
	
}



void save_sysequ() {

	FILE* fp;

	fp = fopen("output.txt","w");
	for (int i=0; i<N; i++)
	{
		for (int j=0; j<N+1; j++)
		{
			if (j == N)
	          fprintf(fp, "%.2f\n", b[i]);
	        else
	          fprintf(fp, "%.2f\t", a[i][j]);
		}
	}
	fclose(fp);
}



void load_sysequ() {
	
	FILE* fp;

    fp = fopen(input_file,"r");
    for (int i=0; i<N; i++) {
      for (int j=0; j<N+1; j++) {
        if (j == N)
          fscanf(fp,"%f\n",&b[i]);
        else
          fscanf(fp,"%f ",&a[i][j]);
      }
    }
    fclose(fp);

}



int main(void) {

	float elapsedTime, bw;
	hipEvent_t start, stop;

	float *d_a;
	float *d_b;
	int *d_i;


	load_sysequ();

	hipMalloc((void**)&d_a, N*N*sizeof(float));
	hipMalloc((void**)&d_b, N*sizeof(float));
	hipMalloc((void**)&d_i, sizeof(int));
	hipMemcpy(d_a, a, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);

	
	// Creating events to estimate execution time
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0); // Starting clock
	bw = 0.0;

	for (int i=0; i<N-1; i++)
	{
		hipMemcpy(d_i, &i, sizeof(int), hipMemcpyHostToDevice);
		add<<<N-i-1,N-i+1>>>(d_a, d_b, d_i);

		bw += (N-i-1)*(N-i+1);
	}

	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(a, d_a, N*N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(b, d_b, N*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_i);

	save_sysequ();

	bw /= elapsedTime*1000000.0;
	printf("Gaussian-Elimination GPU execution time: %7.3f ms, Throughput %6.3f GFLOPS\n", elapsedTime, bw);

	return 0;
}