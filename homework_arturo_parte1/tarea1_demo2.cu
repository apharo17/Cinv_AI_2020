
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 4


char input_file[]= "linsys_4.txt";
float a[N][N];
float b[N];



__global__ void add(float *d_a, float *d_b, int *d_i) {
	float temp;
	int i = *d_i;
	int j = blockIdx.x + i + 1;
	
	temp = d_a[j*N+i]/d_a[i*N+i];
	for (int k=i; k<N; k++)
	{
		d_a[j*N+k] = d_a[j*N+k] - d_a[i*N+k]*temp;
	}
	d_b[j] = d_b[j] - d_b[i]*temp;
}



void save_sysequ() {

	FILE* fp;

	fp = fopen("output.txt","w");
	for (int i=0; i<N; i++)
	{
		for (int j=0; j<N+1; j++)
		{
			if (j == N)
	          fprintf(fp, "%.2f\n", b[i]);
	        else
	          fprintf(fp, "%.2f\t", a[i][j]);
		}
	}
	fclose(fp);
}



void load_sysequ() {
	
	FILE* fp;

    fp = fopen(input_file,"r");
    for (int i=0; i<N; i++) {
      for (int j=0; j<N+1; j++) {
        if (j == N)
          fscanf(fp,"%f\n",&b[i]);
        else
          fscanf(fp,"%f ",&a[i][j]);
      }
    }
    fclose(fp);

}



int main(void) {

	float *d_a;
	float *d_b;
	int *d_i;

	load_sysequ();

	hipMalloc((void**)&d_a, N*N*sizeof(float));
	hipMalloc((void**)&d_b, N*sizeof(float));
	hipMalloc((void**)&d_i, sizeof(int));
	hipMemcpy(d_a, a, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);

	for (int i=0; i<N-1; i++)
	{
		hipMemcpy(d_i, &i, sizeof(int), hipMemcpyHostToDevice);
		add<<<N-i-1,1>>>(d_a, d_b, d_i);
		hipDeviceSynchronize();
		hipMemcpy(a, d_a, N*N*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(b, d_b, N*sizeof(float), hipMemcpyDeviceToHost);		
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_i);

	save_sysequ();

	return 0;
}