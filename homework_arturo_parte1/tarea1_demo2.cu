
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 4


char filename[]= "linsys_4.txt";
float a[N][N];
float b[N];



__global__ void add(float *d_a, float *d_b, int *d_i) {
	float temp;
	int i = *d_i;
	int j = blockIdx.x + i + 1;
	

	//temp = d_a[j][i]/d_a[i][i];
	temp = d_a[j*N+i]/d_a[i*N+i];
	for (int k=i; k<N; k++)
	{
		//d_a[j][k] = d_a[j][k] - d_a[i][k]*temp;
		d_a[j*N+k] = d_a[j*N+k] - d_a[i*N+k]*temp;
	}
	d_b[j] = d_b[j] - d_b[i]*temp;
}



void print_sysequ() {

	for (int i=0; i<N; i++)
	{
		for (int j=0; j<N; j++)
		{
			printf("%.2f\t",a[i][j]);
		}
		printf("%.2f\n",b[i]);
	}
	printf("\n\n");
}



void load_sysequ() {
	
	FILE* fp;

    fp = fopen(filename,"r");
    for (int i=0; i<N; i++) {
      for (int j=0; j<N+1; j++) {
        if (j == N)
          fscanf(fp,"%f\n",&b[i]);
        else
          fscanf(fp,"%f ",&a[i][j]);
      }
    }
    fclose(fp);

}


int main(void) {

	float *d_a;
	float *d_b;
	int *d_i;

	load_sysequ();	
	print_sysequ();

	hipMalloc((void**)&d_a, N*N*sizeof(float));
	hipMalloc((void**)&d_b, N*sizeof(float));
	hipMalloc((void**)&d_i, sizeof(int));
	hipMemcpy(d_a, a, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);

	for (int i=0; i<N-1; i++)
	{
		hipMemcpy(d_i, &i, sizeof(int), hipMemcpyHostToDevice);
		add<<<N-i-1,1>>>(d_a, d_b, d_i);
		hipDeviceSynchronize();
		hipMemcpy(a, d_a, N*N*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(b, d_b, N*sizeof(float), hipMemcpyDeviceToHost);
		print_sysequ();			
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_i);

	return 0;
}