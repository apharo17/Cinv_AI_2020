
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define NPOINTS 128
#define NDIMS 5
#define ITERS 1000

float points_x[NPOINTS][NDIMS];
float points_y[NPOINTS];


double rand_gen() {
   // return a uniformly distributed random value
   return ((double)(rand())+1.)/((double)(RAND_MAX)+1.);
}

double normalRandom() {
   // return a normally distributed random value
   double v1=rand_gen();
   double v2=rand_gen();
   return cos(2*3.14*v2)*sqrt(-2.*log(v1));
}


void print_points() {
	for (int i=0; i<NPOINTS; i++) {
		for (int j=0; j<NDIMS; j++) {
			printf("%.3f\t", points_x[i][j]);
		}
		printf("---> %.3f\n", points_y[i]);
	}
}


void gradientDescent() {
	float eta = 0.01;
	float w[NDIMS];
	float dotprod, cost, temp;
	float gradient[NDIMS];

	for (int i=0; i<NDIMS; i++)
		w[i] = 0;

	
	for (int k=0; k<ITERS; k++) {

		//Calculate the cost
		cost = 0.0;
		for (int i=0; i<NPOINTS; i++) {
			dotprod = 0.0;
			for (int j=0; j<NDIMS; j++) {
				dotprod += w[j]*points_x[i][j];
			}
			cost += (dotprod-points_y[i])*(dotprod-points_y[i]);
		}
		cost /= (1.0*NPOINTS);


		//Calculate the gradient of the cost
		for (int i=0; i<NPOINTS; i++) {
			dotprod = 0.0;
			for (int j=0; j<NDIMS; j++) {
				dotprod += w[j]*points_x[i][j];
			}

			temp = 2.0*(dotprod-points_y[i]);
			
			for (int j=0; j<NDIMS; j++) {
				gradient[j] += temp*points_x[i][j];
			}
		}
		for (int i=0; i<NDIMS; i++) {
			gradient[i] /= (1.0*NPOINTS);
		}


		//Update weights
		for (int i=0; i<NDIMS; i++) {
			w[i] -= eta*gradient[i];
		}


		if (k<10) {
			printf("Iteration %d, Cost = %.4f\n", k, cost);
			printf("Weights:\t");
			for (int i=0; i<NDIMS; i++) {
				printf("%.4f\t", w[i]);
			}
			printf("\n");
		}

	}




}


int main() {
	
	float true_w[NDIMS] = {1, 2, 3, 4, 5};

	srand(time(0));

	float dotprod;
	for (int i=0; i<NPOINTS; i++) {
		for (int j=0; j<NDIMS; j++) {
			points_x[i][j] = (float) normalRandom();
		}
	}
	for (int i=0; i<NPOINTS; i++) {
		dotprod = 0.0;
		for (int j=0; j<NDIMS; j++) {
			dotprod += true_w[j]*points_x[i][j];
		}
		points_y[i] = dotprod + (float)normalRandom()*.2;
	}
	
	//print_points();
	gradientDescent();

	return 0;
}