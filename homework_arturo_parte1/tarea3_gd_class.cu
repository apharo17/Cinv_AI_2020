#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NPOINTS 1024 //It has to be a power of two (from 64 to 1024)
#define NDIMS 2
#define ITERS 1000

float points_x[NPOINTS][NDIMS];
float points_x_copy[NPOINTS][NDIMS];
float points_y[NPOINTS];
float w[NDIMS];

double rand_gen() {
   // return a uniformly distributed random value
   return ((double)(rand())+1.)/((double)(RAND_MAX)+1.);
}

void print_points() {
	for (int i=0; i<NPOINTS; i++) {
		for (int j=0; j<NDIMS; j++) {
			printf("%.3f\t", points_x[i][j]);
		}
		printf("---> %.3f\n", points_y[i]);
	}
}

void normalize (float *x, int n) {
	float square_sum = 0.0;
	for (int i=0; i<n; i++)
		square_sum += x[i]*x[i];
	square_sum = sqrt(square_sum);
	for (int i=0; i<n; i++)
		x[i] /= square_sum;
}


float compute_cost() {
	float cost, dotprod;
	
	cost = 0.0;
	for (int i=0; i<NPOINTS; i++) {
		dotprod = 0.0;
		for (int j=0; j<NDIMS; j++) {
			dotprod += w[j]*points_x[i][j];
		}
		cost += (dotprod-points_y[i])*(dotprod-points_y[i]);
	}
	cost /= (1.0*NPOINTS);

	return cost;
}




__global__ void compute_sigmoid_score(float *d_points_x, float *d_w, float *d_score)
{
    int point_idx = blockIdx.x; //index for point
    int dim_idx = threadIdx.x; //index for dimension
    float res = 0.0;

    extern __shared__ float sdata[];

    sdata[dim_idx] = d_w[dim_idx] * d_points_x[point_idx*blockDim.x + dim_idx];
    __syncthreads();

    if (dim_idx == 0) {
    	for (int k=0; k<NDIMS; k++)
    		res += sdata[k];
    	d_score[point_idx] = 1.0/(1.0 + expf(-res));
    }
}


__global__ void substract_y(float *d_score, float *d_points_y)
{
    int i = threadIdx.x; //index for point

    d_score[i] -=  d_points_y[i];
}


__global__ void scale_x(float *d_score, float *d_points_x, float *d_points_x_copy)
{
    int i = blockIdx.x; //index for point
    int j = threadIdx.x; //index for element

    d_points_x_copy[i*blockDim.x+j] = 2.0*d_score[i]*d_points_x[i*blockDim.x+j];
}


__global__ void compute_gradient(float *d_points_x_copy, float *d_partial_grad)
{
	int block_idx = blockIdx.x; //index for block
	int point_idx = threadIdx.x; //index for point
	int dim_idx = threadIdx.y; //index for dimension

    __shared__ float sdata[64][NDIMS];

    sdata[point_idx][dim_idx] = d_points_x_copy[(block_idx+1)*point_idx*NDIMS + dim_idx];
    __syncthreads();

	for (unsigned int s=1; s < blockDim.x; s *= 2) {
		int index = 2 * s * point_idx;
		if (index < blockDim.x) {
			sdata[index][dim_idx] += sdata[index + s][dim_idx];
		}
		__syncthreads();
	}

	if (point_idx == 0)
		d_partial_grad[block_idx*NDIMS + dim_idx] = sdata[0][dim_idx]/64.0;
}




void gradientDescent() {
	
	float elapsedTime, bw;
	hipEvent_t start, stop;

	float eta = 0.01;
	float gradient[NDIMS];
	float partial_grad[NPOINTS/64][NDIMS];

	for (int i=0; i<NDIMS; i++) {
		w[i] = 0.;
		gradient[i] = 0.;
	}

	float *d_points_x, *d_w, *d_score, *d_points_x_copy, *d_points_y, *d_partial_grad;
	hipMalloc((void**)&d_points_x, sizeof(float)*NPOINTS*NDIMS);
	hipMalloc((void**)&d_w, sizeof(float)*NDIMS);
	hipMalloc((void**)&d_score, sizeof(float)*NPOINTS);
	hipMalloc((void**)&d_points_x_copy, sizeof(float)*NPOINTS*NDIMS);
	hipMalloc((void**)&d_points_y, sizeof(float)*NPOINTS);
	hipMalloc((void**)&d_partial_grad, sizeof(float)*(NPOINTS/64)*NDIMS);

	hipMemcpy(d_points_x, points_x, sizeof(float)*NPOINTS*NDIMS, hipMemcpyHostToDevice);
	hipMemcpy(d_points_y, points_y, sizeof(float)*NPOINTS, hipMemcpyHostToDevice);


	// Creating events to estimate execution time
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0); // Starting clock
	bw = 0.0;


	for (int k=0; k<ITERS; k++) {

		hipMemcpy(d_w, w, sizeof(float)*NDIMS, hipMemcpyHostToDevice);
		
		compute_sigmoid_score<<<NPOINTS,NDIMS,NDIMS*sizeof(float)>>>(d_points_x, d_w, d_score);

		substract_y<<<1,NPOINTS>>>(d_score, d_points_y);
		
		scale_x<<<NPOINTS,NDIMS>>>(d_score, d_points_x, d_points_x_copy);

		dim3 threads(64,NDIMS);
		compute_gradient<<<(NPOINTS/64),threads>>>(d_points_x_copy, d_partial_grad);

		hipMemcpy(partial_grad, d_partial_grad, sizeof(float)*(NPOINTS/64)*NDIMS, hipMemcpyDeviceToHost);

		for (int i=0; i<(NPOINTS/64); i++) {
			for (int j=0; j<NDIMS; j++) {
				gradient[j] += partial_grad[i][j];
			}
		}
		for (int i=0; i<NDIMS; i++) {
			gradient[i] /= (NPOINTS/64.0);
		}

		//Update weights
		for (int i=0; i<NDIMS; i++) {
			w[i] -= eta*gradient[i];
		}

		normalize(w, NDIMS);
	}

	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

	hipFree(d_points_x);
	hipFree(d_w);
	hipFree(d_score);
	hipFree(d_points_x_copy);
	hipFree(d_points_y);
	hipFree(d_partial_grad);

	printf("Total of points: %d\n",NPOINTS);
	printf("Weights: ");
	for (int i=0; i<NDIMS; i++) {
		printf("%.4f\t", w[i]);
	}
	printf("\n");
	printf("Final cost: %.4f\n", compute_cost());

	bw = NPOINTS*NDIMS*ITERS/(elapsedTime*1000.0);
	printf("Gradient descent for regression, GPU execution time: %7.3f ms, Throughput %6.3f MFLOPS\n", elapsedTime, bw);
	
}




int main() {
	
	float true_w[NDIMS] = {-1, 1};
	normalize(true_w, NDIMS);


	srand(time(0));

	//Create dataset
	float dotprod;
	float point_x[NDIMS];
	for (int i=0; i<NPOINTS; i++) {
		for (int j=0; j<NDIMS; j++) {
			point_x[j] = (float) rand_gen();
		}
		normalize(point_x,NDIMS);
		for (int j=0; j<NDIMS; j++) {
			points_x[i][j] = point_x[j];
		}
	}
	for (int i=0; i<NPOINTS; i++) {
		dotprod = 0.0;
		for (int j=0; j<NDIMS; j++) {
			dotprod += true_w[j]*points_x[i][j];
		}
		points_y[i] = (dotprod>0) ? 1.0 : 0.0;
	}
	
	gradientDescent();

	return 0;
}